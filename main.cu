﻿#include <stdio.h>
#include <functional>

#include "hip/hip_runtime.h"
#include ""
#include "glut_functions.cuh"
#include "kernels.cuh"

namespace glf = glut_functions;

int main(int argc, char** argv)
{
   glf::window = new Window(data::BOIDS_COUNT);

   glutInit(&argc, argv);
   glutInitDisplayMode(GLUT_RGB);
   glutInitWindowSize(data::WIDTH, data::HEIGHT);
   glutCreateWindow("Boids");

   glShadeModel(GL_FLAT);
   glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

   glutDisplayFunc(glf::display);
   glutReshapeFunc(glf::reshape);
   glutKeyboardFunc(glf::keyboardLetters);
   glutMouseFunc(glf::mouse);
   glutPassiveMotionFunc(glf::mousePassive);
   atexit(glf::exitingFunction);
   glutTimerFunc(0, glf::onTimer, 0);
   glutMainLoop();

   return 0;
}