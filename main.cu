﻿#include <stdio.h>
#include <functional>

#include "hip/hip_runtime.h"
#include ""
#include "glut_functions.h"
#include "window.h"

namespace glf = glut_functions;

int main(int argc, char** argv)
{
   glutInit(&argc, argv);
   glutInitDisplayMode(GLUT_RGB);
   glutInitWindowSize(Window::WIDTH, Window::HEIGHT);
   glutCreateWindow("Boids");

   glShadeModel(GL_FLAT);
   glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

   glutDisplayFunc(glf::display);
   glutReshapeFunc(glf::reshape);
   glutKeyboardFunc(glf::keyboardLetters);
   glutMouseFunc(glf::mouse);
   glutPassiveMotionFunc(glf::mousePassive);
   atexit(glf::exitingFunction);
   glutTimerFunc(0, glf::onTimer, 0);

   glf::window = new Window(10);

   glutMainLoop();

   return 0;
}