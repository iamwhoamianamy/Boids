#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "help_functions.cuh"
#include "window.cuh"

namespace hlp = help_functions;

__global__ void makeMetaSpheres(const Boid* boids, float* canvas)
{
   __shared__ uint boid_id;

   __shared__ int start_x;
   __shared__ int start_y;

   if(threadIdx.x == 0 && threadIdx.y == 0)
   {
      boid_id = blockIdx.x;
      start_x = boids[boid_id].pos.x - data::HALF_IMPACT;
      start_y = boids[boid_id].pos.y - data::HALF_IMPACT;
   }

   __syncthreads();

   int offset_y = threadIdx.y;

   while(offset_y < data::IMPACT_WIDTH)
   {
      int offset_x = threadIdx.x;

      while(offset_x < data::IMPACT_WIDTH)
      {
         int x = start_x + offset_x;
         int y = start_y + offset_y;

         if(0 <= x && x < data::WIDTH &&
            0 <= y && y < data::HEIGHT)
         {
            float to_add = data::META_RADIUS *
               hlp::fisqrt((x - boids[boid_id].pos.x) * (x - boids[boid_id].pos.x) +
                           (y - boids[boid_id].pos.y) * (y - boids[boid_id].pos.y));
            //float* cell = ;

            //if(*cell + to_add <= 0.5f)
               atomicAdd(&canvas[x + y * data::WIDTH], to_add);
            //atomicAdd(&canvas[x + y * data::WIDTH], 0.5f);
         }

         __syncthreads();

         offset_x += data::IMPACT_THREAD_WIDTH;
      }
      offset_y += data::IMPACT_THREAD_WIDTH;
   }
}

__global__ void makeMetaSpheresFull(const Boid* boids, float* canvas)
{
   __shared__ uint boid_id;

   if(threadIdx.x == 0 && threadIdx.y == 0)
   {
      boid_id = blockIdx.x;
   }

   __syncthreads();

   int y = threadIdx.y;

   while(y < data::HEIGHT)
   {
      int x = threadIdx.x;

      while(x < data::WIDTH)
      {
         float to_add = data::META_RADIUS *
            hlp::fisqrt((x - boids[boid_id].pos.x) * (x - boids[boid_id].pos.x) +
                        (y - boids[boid_id].pos.y) * (y - boids[boid_id].pos.y));

         atomicAdd(&canvas[x + y * data::WIDTH], to_add);

         __syncthreads();

         x += data::IMPACT_THREAD_WIDTH;
      }
      y += data::IMPACT_THREAD_WIDTH;
   }
}

using namespace data;

__global__ void initBoids(Boid* boids, const int boidsCount)
{
   th_id i = threadIdx.x + blockIdx.x * blockDim.x;

   if(i < boidsCount)
   {
      const float step = 360.0f / boidsCount;
      const float ring_radius = WIDTH / 4;

      Vec pos = Vec(WIDTH / 2 + ring_radius * cos(i * RAD * step),
                    HEIGHT / 2 + ring_radius * sin(i * RAD * step));
      Vec vel = Vec(WIDTH / 2 - pos.x, HEIGHT / 2 - pos.y);

      vel.Limit(0.5f + 2.0f * i / boidsCount);
      boids[i] = Boid(pos, vel);
   }
}

template <class T>
__global__ void clearArray<T>(T* arr)
{
   th_id x = threadIdx.x + blockIdx.x * blockDim.x;
   th_id y = threadIdx.y + blockIdx.y * blockDim.y;
   th_id offset = x + y * blockDim.x * gridDim.x;

   arr[offset] = 0.0f;
}

__host__ void dummyKernelsTemplate()
{
   clearArray<float><<<0, 0>>>(NULL);
}

__device__ void setPixel(uchar* ptr, int offset, const uchar value)
{
   offset *= 3;
   ptr[offset + 0] = value;
   ptr[offset + 1] = value;
   ptr[offset + 2] = value;
}

__global__ void floatToColor(const float* values, uchar* colors)
{
   th_id x = threadIdx.x + blockIdx.x * blockDim.x;
   th_id y = threadIdx.y + blockIdx.y * blockDim.y;
   th_id offset = x + y * blockDim.x * gridDim.x;

   if(offset < GRID_SIZE)
   {
      uchar value;

      if(values[offset] < 0.5f)
         setPixel(colors, offset, 0);
      else
         setPixel(colors, offset, 255);

      //setPixel(colors, offset, hlp::min<uchar>(100, values[offset] * 255));
      //setPixel(colors, offset, 100);
   }
}

__global__ void moveBoids(Boid* boids, const int boidsCount)
{
   th_id i = threadIdx.x + blockIdx.x * blockDim.x;

   if(i < boidsCount)
   {
      boids[i].UpdatePosition(data::WIDTH, data::HEIGHT);
   }
}
