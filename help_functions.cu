#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "help_functions.cuh"

__device__ float help_functions::fisqrt(const float number)
{
   long i;
   float x2, y;
   const float threehalfs = 1.5F;

   x2 = number * 0.5F;
   y = number;
   i = *(long*)&y;
   i = 0x5f3759df - (i >> 1);
   y = *(float*)&i;
   y = y * (threehalfs - (x2 * y * y));

   return y;
}

template<class T>
__device__ T help_functions::max(T t1, T t2)
{
   return t1 < t2 ? t2 : t1;
}

template<class T>
__device__ T help_functions::min(T t1, T t2)
{
   return t1 < t2 ? t1 : t2;
}

typedef unsigned char uchar;

__device__ void dummyHelpTemplate()
{
   help_functions::max<float>(0.0f, 0.0f);
   help_functions::min<float>(0.0f, 0.0f);
   help_functions::min<uchar>(0, 0);

   help_functions::max<int>(0, 0);
   help_functions::min<int>(0, 0);
   help_functions::max<uchar>(0, 0);


}
