#include "hip/hip_runtime.h"
#include ""
#include "boid.cuh"

__device__ Boid::Boid() : pos(), vel() {}

__device__ Boid::Boid(const Vec& pos, const Vec& vel) : pos(pos), vel(vel)
{
}

__device__ Boid::Boid(float px, float py, float vx, float vy) : pos(px, py), vel(vx, vy)
{
}

__device__ void Boid::UpdatePosition(const float width, const float height)
{
   pos += vel;

   if(pos.x < 0)
   {
      vel.x *= -1;
      pos.x = 0;
   }
   else
   {
      if(pos.x >= width)
      {
         vel.x *= -1;
         pos.x = width - 1;
      }
   }

   if(pos.y < 0)
   {
      vel.y *= -1;
      pos.y = 0;
   }
   else
   {
      if(pos.y >= height)
      {
         vel.y *= -1;
         pos.y = height - 1;
      }
   }
}